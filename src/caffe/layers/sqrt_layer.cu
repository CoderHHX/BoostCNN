#include "hip/hip_runtime.h"
// Sqrt neuron activation function layer.
// Adapted from ReLU layer code written by Yangqing Jia

#include <algorithm>
#include <vector>

#include "caffe/layers/sqrt_layer.hpp"

#define EPS ((Dtype) 1e-10)

namespace caffe {

template <typename Dtype>
__global__ void SqrtForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0? sqrt(in[index]): -sqrt(-in[index]);
  }
}

template <typename Dtype>
void SqrtLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SqrtForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void SqrtBackward(const int n, const Dtype* in_diff,
    const Dtype* out_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    // 1e-10 is added to prevent division by zero
    out_diff[index] = 0.5 * in_diff[index] / (EPS + abs(out_data[index]));
  }
}

template <typename Dtype>
void SqrtLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    SqrtBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, top_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SqrtLayer);


}  // namespace caffe
